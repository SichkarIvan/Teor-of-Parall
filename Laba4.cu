#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <sstream>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define ITER_TO_UPDATE 250

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция изменения матрицы
__global__ void step(double* A, double* A_new, size_t size) {
	size_t i = blockIdx.x + 1, j = threadIdx.x + 1;
	A_new[i * size + j] = 0.25 * (A[i * size + j - 1] + A[(i - 1) * size + j] + A[(i + 1) * size + j] + A[i * size + j + 1]);	
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция востановления границ матрицы
__global__ void restore(double* A, int size){
	size_t i = threadIdx.x;
	A[i] = 10.0 + i * 10.0 / (size - 1);
	A[i * size] = 10.0 + i * 10.0 / (size - 1);
	A[size - 1 + i * size] = 20.0 + i * 10.0 / (size - 1);
	A[size * (size - 1) + i] = 20.0 + i * 10.0 / (size - 1);
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция разницы матриц
__global__ void subtraction(double* A, double* A_new) {
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	A_new[idx] = A[idx] - A_new[idx];
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Значения по умодчанию
double eps = 1E-6;
int size = 512;
int iter_max = 1E6;

int main(int argc, char** argv) {
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Получение значений из командной строки
    for(int arg = 0; arg < argc; arg++){ 
        std::stringstream stream;
        if(strcmp(argv[arg], "-error") == 0){
            stream << argv[arg+1];
            stream >> eps;
        }
        else if(strcmp(argv[arg], "-iter") == 0){
            stream << argv[arg+1];
            stream >> iter_max;
        }
        else if(strcmp(argv[arg], "-size") == 0){
            stream << argv[arg+1];
            stream >> size;
        }
    }

	size_t totalSize = size * size;

	std::cout << "Settings: " << "\n\tMin error: " << eps << "\n\tMax iteration: " << iter_max << "\n\tSize: " << size << "x" << size << std::endl;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Выделения памяти
	double* A = new double[totalSize];

	hipSetDevice(0);

	hipStream_t stream;
	hipStreamCreate(&stream);
	hipGraph_t graph;
	hipGraphExec_t graph_instance;

	double* A_Device, *A_new_Device, *deviceError, *tempStorage = NULL;
	size_t tempStorageSize = 0;

	hipMalloc(&A_Device, sizeof(double) * totalSize);
	hipMalloc(&A_new_Device, sizeof(double) * totalSize);
	hipMalloc(&deviceError, sizeof(double));

	restore<<<1, size>>>(A_Device, size);
	hipMemcpy(A_new_Device, A_Device, sizeof(double) * totalSize, hipMemcpyDeviceToDevice);

	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_new_Device, deviceError, totalSize, stream);
	hipMalloc(&tempStorage, tempStorageSize);
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Создание графа
	hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

	for (size_t i = 0; i < ITER_TO_UPDATE; i += 2) {
		step<<<size - 2, size - 2, 0, stream>>>(A_Device, A_new_Device, size);
		step<<<size - 2, size - 2, 0, stream>>>(A_new_Device, A_Device, size);
	}
	subtraction<<<size, size, 0, stream>>>(A_Device, A_new_Device);
	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_new_Device, deviceError, totalSize, stream);
	restore<<<1, size, 0, stream>>>(A_new_Device, size);

	hipStreamEndCapture(stream, &graph);
	hipGraphInstantiate(&graph_instance, graph, NULL, NULL, 0);
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Основной цикл
	int iter = 0; 
	double error = 1.0;
	while(iter < iter_max && error > eps) {
		hipGraphLaunch(graph_instance, stream);
		hipMemcpy(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost);
		iter += ITER_TO_UPDATE;
	}
	std::cout << "Result:\n\tIter: " << iter << "\n\tError: " << error << std::endl;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Чистка памяти
	hipFree(A_Device);
	hipFree(A_new_Device);
	hipFree(tempStorage);
	hipGraphDestroy(graph);
	hipStreamDestroy(stream);
	delete[] A;
	return 0;
}